#include "hip/hip_runtime.h"
#include "libcumat.h"

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n" , __FILE__ , __LINE__);		 \
    exit(EXIT_FAILURE);}} while(0)

namespace Cumat
{

hipblasHandle_t cublas_handle;
std::unordered_map<std::string, char *> kernel_cache;

void init(void)
{
	checkCudaErrors(hipblasCreate(&Cumat::cublas_handle));
}

void end(void)
{
	checkCudaErrors(hipblasDestroy(Cumat::cublas_handle));
	for(std::pair<std::string, char *> iter : kernel_cache)
		delete iter.second;
}

//----------------------------------------------
// Private methods
//----------------------------------------------

template<typename T>
template<class F>
void Matrix<T>::elementMathOp(Matrix<T> &src, Matrix<T> &dst, const F &func)
{
	if (src.rows_ == 0 || src.cols_ == 0)
		return;

	thrust::transform(src.data_.begin(), src.data_.end(), dst.data_.begin(), func);
}

template<>
const std::string Matrix<float>::type(void) const
{
	return "float";
}

template<>
const std::string Matrix<double>::type(void) const
{
	return "double";
}

//----------------------------------------------
// CUDA Library Wrappers
//----------------------------------------------

template<>
void Matrix<float>::curandGenerateRandom(hiprandGenerator_t &generator, float *output, size_t size)
{
	CURAND_CALL(hiprandGenerateUniform(generator, output, size));
}

template<>
void Matrix<double>::curandGenerateRandom(hiprandGenerator_t &generator, double *output, size_t size)
{
	CURAND_CALL(hiprandGenerateUniformDouble(generator, output, size));
}

template<>
void Matrix<float>::cublasTranspose(hipblasHandle_t &handle, const int rows, const int cols, const float *alpha, const float *in_mat, const float *beta, float *out_mat)
{
	checkCudaErrors(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, rows, cols, alpha, in_mat, cols, beta, in_mat, cols, out_mat, rows));
}

template<>
void Matrix<double>::cublasTranspose(hipblasHandle_t &handle, const int rows, const int cols, const double *alpha, const double *in_mat, const double *beta, double *out_mat)
{
	checkCudaErrors(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, rows, cols, alpha, in_mat, cols, beta, in_mat, cols, out_mat, rows));
}

template<>
void Matrix<float>::cublasAxpy(hipblasHandle_t &handle, const int size, const float alpha, const float *x, const int incx, float *y, const int incy)
{
	checkCudaErrors(hipblasSaxpy(handle, size, &alpha, x, incx, y, incy));
}

template<>
void Matrix<double>::cublasAxpy(hipblasHandle_t &handle, const int size, const double alpha, const double *x, const int incx, double *y, const int incy)
{
	checkCudaErrors(hipblasDaxpy(handle, size, &alpha, x, incx, y, incy));
}

template<>
void Matrix<float>::cublasScal(hipblasHandle_t &handle, const int size, const float alpha, float *x, int incx)
{
	checkCudaErrors(hipblasSscal(handle, size, &alpha, x, incx));
}

template<>
void Matrix<double>::cublasScal(hipblasHandle_t &handle, const int size, const double alpha, double *x, int incx)
{
	checkCudaErrors(hipblasDscal(handle, size, &alpha, x, incx));
}

template<>
void Matrix<float>::cublasGemm(hipblasHandle_t &handle, int m, int n, int k, const float alpha, const float *A, int lda, const float *B, int ldb, const float beta, float *C, int ldc)
{
	checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc));
}

template<>
void Matrix<double>::cublasGemm(hipblasHandle_t &handle, int m, int n, int k, const double alpha, const double *A, int lda, const double *B, int ldb, const double beta, double *C, int ldc)
{
	checkCudaErrors(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc));
}

template<>
void Matrix<float>::cublasNorm(hipblasHandle_t &handle, int size, const float *x, int incx, float *result)
{
	checkCudaErrors(hipblasSnrm2(handle, size, x, incx, result));
}

template<>
void Matrix<double>::cublasNorm(hipblasHandle_t &handle, int size, const double *x, int incx, double *result)
{
	checkCudaErrors(hipblasDnrm2(handle, size, x, incx, result));
}

//----------------------------------------------
// Public methods
//----------------------------------------------

template<typename T>
Matrix<T>::Matrix(const size_t rows, const size_t cols):
	rows_(rows),
	cols_(cols),
	data_(rows_ * cols_),
	id_("v")
{
	if (rows == 0 || cols == 0) {
		rows_ = 0;
		cols_ = 0;
	}

	data_ptr_ = (hipDeviceptr_t)thrust::raw_pointer_cast(data_.data());
}

template<typename T>
Matrix<T>::Matrix(const size_t rows, const size_t cols, const T val):
	rows_(rows),
	cols_(cols),
	data_(rows_ * cols_, val),
	id_("v")
{
	if (rows == 0 || cols == 0) {
		rows_ = 0;
		cols_ = 0;
	}

	data_ptr_ = (hipDeviceptr_t)thrust::raw_pointer_cast(data_.data());
}

template<typename T>
Matrix<T>::Matrix(void):
	rows_(0),
	cols_(0),
	data_(rows_ * cols_),
	id_("v")
{
	data_ptr_ = (hipDeviceptr_t)thrust::raw_pointer_cast(data_.data());
}

template<typename T>
std::string Matrix<T>::buildKernel(std::string &params, int &num, std::vector<void *> &args, const bool &transpose) const
{
	std::string id_num = std::to_string(num++);
	params += (", " + this->type() + " *v" + id_num);
	args.push_back((void *)&data_ptr_);
	return id_ + id_num + ((transpose) ? "[x * rows + y]" : "[y * cols + x]");
}

template<typename T>
const Matrix<T>& Matrix<T>::eval(void) const
{
	return *this;
}

template<typename T>
size_t Matrix<T>::rows(void) const
{
	return rows_;
}

template<typename T>
size_t Matrix<T>::cols(void) const
{
	return cols_;
}

template<typename T>
size_t Matrix<T>::size(void) const
{
	return rows_ * cols_;
}

template<typename T>
void Matrix<T>::resize(size_t rows, size_t cols)
{
	if (rows_ == rows && cols_ == cols) return;

	if (rows == 0 || cols == 0) {
		rows = 0;
		cols = 0;
	}

	if (rows_ * cols_ != rows * cols) {
		data_.resize(rows * cols);
		data_ptr_ = (hipDeviceptr_t)thrust::raw_pointer_cast(data_.data());
	}

	rows_ = rows;
	cols_ = cols;
}

template<typename T>
void Matrix<T>::set(const size_t row, const size_t col, const T val)
{
	assert(row < rows_ && col < cols_);
	data_[row * cols_ + col] = val;
}

template<typename T>
void Matrix<T>::set(const size_t idx, const T val)
{
	assert(idx < rows_ * cols_);
	data_[idx] = val;
}

template<typename T>
void Matrix<T>::swap(Matrix<T> &mat)
{
	if (&mat == this) return;
	std::swap(rows_, mat.rows_);
	std::swap(cols_, mat.cols_);
	std::swap(data_ptr_, mat.data_ptr_);
	data_.swap(mat.data_);
}

template<typename T>
void Matrix<T>::fill(const T val)
{
	thrust::fill(data_.begin(), data_.end(), val);
}

template<typename T>
void Matrix<T>::zero(void)
{
	Matrix<T>::fill(0);
}

template<typename T>
void Matrix<T>::rand(const T min, const T max)
{
	assert(max > min);

	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());
	Matrix<T>::curandGenerateRandom(prng, thrust::raw_pointer_cast(data_.data()), rows_ * cols_);

	(*this *= (max - min)) += min;
}

template<typename T>
Matrix<T> Matrix<T>::random(const size_t rows, const size_t cols, const T min, const T max)
{
	assert(max > min);
	Matrix<T> mat(rows, cols);
	mat.rand(min, max);
	return mat;
}

template<typename T>
void Matrix<T>::transpose(void)
{
	thrust::device_vector<T> temp(cols_ * rows_);

	T alpha = 1.0;
	T beta = 0;

	T *A = thrust::raw_pointer_cast(data_.data());
	T *B = thrust::raw_pointer_cast(temp.data());

	Matrix<T>::cublasTranspose(Cumat::cublas_handle, rows_, cols_, &alpha, A, &beta, B);

	data_.swap(temp);
	data_ptr_ = (hipDeviceptr_t)thrust::raw_pointer_cast(data_.data());
	this->resize(cols_, rows_);
}

template<typename T>
Matrix<T>& Matrix<T>::transpose(Matrix<T> &mat)
{
	assert(&mat != this);

	if (mat.rows_ != cols_ && mat.cols_ != rows_)
		mat.resize(cols_, rows_);

	T alpha = 1.0;
	T beta = 0;

	T *A = thrust::raw_pointer_cast(data_.data());
	T *B = thrust::raw_pointer_cast(mat.data_.data());

	Matrix<T>::cublasTranspose(Cumat::cublas_handle, rows_, cols_, &alpha, A, &beta, B);

	return mat;
}

template<typename T>
Matrix<T> Matrix<T>::mmul(const Matrix<T> &mat)
{
	assert(cols_ == mat.rows_);

	Matrix<T> outmat(rows_, mat.cols_);

	if (outmat.rows_ == 0 || outmat.cols_ == 0)
		return outmat;

	outmat.mmul(*this, mat, 0);

	return outmat;
}

template<typename T>
Matrix<T>& Matrix<T>::mmul(const Matrix<T> &lhs, const Matrix<T> &rhs, const T beta)
{
	size_t outrows = lhs.rows_;
	size_t outcols = rhs.cols_;

	assert(lhs.cols_ == rhs.rows_ && this != &lhs && this != &rhs);

	if (rows_ != outrows || cols_ != outcols)
		this->resize(outrows, outcols);

	const T *A = thrust::raw_pointer_cast(lhs.data_.data());
	const T *B = thrust::raw_pointer_cast(rhs.data_.data());
	T *C = thrust::raw_pointer_cast(data_.data());
	
	// Use cublas<t>gemm() to perform C = alpha * A * B + beta * C
	// where alpha = 1, A = data_, B = mat, beta = 0, and C = outmat
	Matrix<T>::cublasGemm(Cumat::cublas_handle, rhs.cols_, lhs.rows_, lhs.cols_, 1.0, B, rhs.cols_, A, lhs.cols_, beta, C, rhs.cols_);

	return *this;
}

template<typename T>
T Matrix<T>::sum(void)
{
	return thrust::reduce(data_.begin(), data_.end());
}

template<typename T>
T Matrix<T>::norm(void)
{
	const T *X = thrust::raw_pointer_cast(data_.data());
	T result;

	Matrix<T>::cublasNorm(Cumat::cublas_handle, rows_ * cols_, X, 1, &result);

	return result;
}

template<typename T>
T Matrix<T>::maxElement(void)
{
	typename thrust::device_vector<T>::iterator iter = thrust::max_element(data_.begin(), data_.end());
	return *iter;
}

template<typename T>
int Matrix<T>::maxIndex(void)
{
	typename thrust::device_vector<T>::iterator iter = thrust::max_element(data_.begin(), data_.end());
	return iter - data_.begin();
}

template<typename T>
T Matrix<T>::minElement(void)
{
	typename thrust::device_vector<T>::iterator iter = thrust::min_element(data_.begin(), data_.end());
	return *iter;
}

template<typename T>
int Matrix<T>::minIndex(void)
{
	typename thrust::device_vector<T>::iterator iter = thrust::min_element(data_.begin(), data_.end());
	return iter - data_.begin();
}

//----------------------------------------------
// Element-Wise Math Operations
// *this = op(mat)
//----------------------------------------------

template<typename T>
Matrix<T>& Matrix<T>::abs(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::abs<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::inverse(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::inverse<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::clip(Matrix<T> &mat, const T min, const T max)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::clip<T>(min, max));
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::exp(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::exp<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::exp10(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::exp10<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::exp2(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::exp2<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::log(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::log<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::log1p(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::log1p<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::log10(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::log10<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::log2(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::log2<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::pow(Matrix<T> &mat, const T n)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::pow<T>(n));
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::square(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::square<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::sqrt(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::sqrt<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::rsqrt(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::rsqrt<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::cube(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::cube<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::cbrt(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::cbrt<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::rcbrt(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::rcbrt<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::sin(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::sin<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::cos(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::cos<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::tan(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::tan<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::asin(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::asin<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::acos(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::acos<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::atan(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::atan<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::sinh(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::sinh<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::cosh(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::cosh<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::tanh(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::tanh<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::asinh(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::asinh<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::acosh(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::acosh<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::atanh(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::atanh<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::sigmoid(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::sigmoid<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::ceil(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::ceil<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::floor(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::floor<T>());
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::round(Matrix<T> &mat)
{
	Matrix<T>::elementMathOp(mat, *this, MathOp::round<T>());
	return *this;
}

//----------------------------------------------
// In-Place Element-Wise Math Operations
// *this = op(*this)
//----------------------------------------------

template<typename T>
Matrix<T>& Matrix<T>::abs(void)
{
	this->abs(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::inverse(void)
{
	this->inverse(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::clip(const T min, const T max)
{
	this->clip(*this, min, max);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::exp(void)
{
	this->exp(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::exp10(void)
{
	this->exp10(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::exp2(void)
{
	this->exp2(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::log(void)
{
	this->log(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::log1p(void)
{
	this->log1p(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::log10(void)
{
	this->log10(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::log2(void)
{
	this->log2(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::pow(const T n)
{
	this->pow(*this, n);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::square(void)
{
	this->square(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::sqrt(void)
{
	this->sqrt(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::rsqrt(void)
{
	this->rsqrt(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::cube(void)
{
	this->cube(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::cbrt(void)
{
	this->cbrt(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::rcbrt(void)
{
	this->rcbrt(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::sin(void)
{
	this->sin(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::cos(void)
{
	this->cos(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::tan(void)
{
	this->tan(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::asin(void)
{
	this->asin(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::acos(void)
{
	this->acos(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::atan(void)
{
	this->atan(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::sinh(void)
{
	this->sinh(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::cosh(void)
{
	this->cosh(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::tanh(void)
{
	this->tanh(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::asinh(void)
{
	this->asinh(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::acosh(void)
{
	this->acosh(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::atanh(void)
{
	this->atanh(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::sigmoid(void)
{
	this->sigmoid(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::ceil(void)
{
	this->ceil(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::floor(void)
{
	this->floor(*this);
	return *this;
}

template<typename T>
Matrix<T>& Matrix<T>::round(void)
{
	this->round(*this);
	return *this;
}

//----------------------------------------------
// Operator Overloads
//----------------------------------------------

// -------------- Assignment --------------
template<typename T>
Matrix<T>& Matrix<T>::operator=(const Matrix<T> &rhs)
{
	if (&rhs == this)
		return *this;

	if (rows_ != rhs.rows_ || cols_ != rhs.cols_)
		this->resize(rhs.rows_, rhs.cols_);

	thrust::copy(rhs.data_.begin(), rhs.data_.end(), data_.begin());

	return *this;
}

// -------------- Accessor --------------
template<typename T>
T Matrix<T>::operator()(const size_t row, const size_t col) const
{
	assert(row < rows_ && col < cols_);
	return data_[row * cols_ + col];
}

template<typename T>
T Matrix<T>::operator()(const size_t idx) const
{
	assert(idx < rows_ * cols_);
	return data_[idx];
}

// -------------- Matrix Multiplication --------------
// template<typename T>
// Matrix<T> Matrix<T>::operator^(const Matrix<T> &rhs)
// {
	// return (*this).mmul(rhs);
// }

// -------------- Scalar Addition --------------
template<typename T>
Matrix<T>& Matrix<T>::operator+=(const T val)
{
	T *scalar = nullptr;

	// Create a temporary buffer on the device for the single scalar value
	checkCudaErrors(hipMalloc((void **)&scalar, sizeof(T)));
	checkCudaErrors(hipMemcpy(scalar, &val, sizeof(T), hipMemcpyHostToDevice));

	// use cuBLAS saxpy to do y = alpha * x + y where alpha = 1, x = val, and y = data_
	Matrix<T>::cublasAxpy(Cumat::cublas_handle, rows_ * cols_, 1.0, scalar, 0, thrust::raw_pointer_cast(data_.data()), 1);

	checkCudaErrors(hipFree(scalar));

	return *this;
}

// -------------- Matrix Addition --------------
template<typename T>
Matrix<T>& Matrix<T>::operator+=(const Matrix<T> &rhs)
{
	assert(rows_ == rhs.rows_ && cols_ == rhs.cols_);

	const T *X = thrust::raw_pointer_cast(rhs.data_.data());
	T *Y = raw_pointer_cast(data_.data());

	// use cuBLAS saxpy to do y = alpha * x + y where alpha = 1, x = rhs, and y = data_
	Matrix<T>::cublasAxpy(Cumat::cublas_handle, rows_ * cols_, 1.0, X, 1, Y, 1);

	return *this;
}

// -------------- Scalar Subtraction --------------
template<typename T>
Matrix<T>& Matrix<T>::operator-=(const T val)
{
	*this += -val;
	return *this;
}

// -------------- Matrix Subtraction --------------
template<typename T>
Matrix<T>& Matrix<T>::operator-=(const Matrix<T> &rhs)
{
	assert(rows_ == rhs.rows_ && cols_ == rhs.cols_);

	const T *X = thrust::raw_pointer_cast(rhs.data_.data());
	T *Y = thrust::raw_pointer_cast(data_.data());

	// use cuBLAS saxpy to do y = alpha * x + y where alpha = -1, x = rhs, and y = data_
	Matrix<T>::cublasAxpy(Cumat::cublas_handle, rows_ * cols_, -1.0, X, 1, Y, 1);

	return *this;
}

// -------------- Scalar Multiplication --------------
template<typename T>
Matrix<T>& Matrix<T>::operator*=(const T val)
{
	// Use cublas<t>scal to do x = alpha * x where alpha = val and x = data_
	Matrix<T>::cublasScal(Cumat::cublas_handle, rows_ * cols_, val, thrust::raw_pointer_cast(data_.data()), 1);

	return *this;
}

// -------------- Matrix Multiplication (element-wise) --------------
template<typename T>
Matrix<T>& Matrix<T>::operator*=(const Matrix<T> &rhs)
{
	assert(rows_ == rhs.rows_ && cols_ == rhs.cols_);
	thrust::transform(data_.begin(), data_.end(), rhs.data_.begin(), data_.begin(), thrust::multiplies<T>());
	return *this;
}

// -------------- Scalar Division (element-wise) --------------
template<typename T>
Matrix<T>& Matrix<T>::operator/=(const T val)
{
	*this *= (1.0 / val);
	return *this;
}

// -------------- Matrix Division (element-wise) --------------
template<typename T>
Matrix<T>& Matrix<T>::operator/=(const Matrix<T> &rhs)
{
	assert(rows_ == rhs.rows_ && cols_ == rhs.cols_);
	thrust::transform(data_.begin(), data_.end(), rhs.data_.begin(), data_.begin(), thrust::divides<T>());
	return *this;
}
};

// Template explicit instantiation
template class Cumat::Matrix<float>;
template class Cumat::Matrix<double>;
